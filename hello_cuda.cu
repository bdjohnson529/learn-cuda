/*------------------
Simple CUDA Example
prints hello from threads and performs vector addition on the GPU.
Demonstrates kernel launch and device/host memory management.

Two kernels:
* hello_kernel: prints thread/block info from the GPU.
* vector_add: adds two float arrays on the GPU.

Usage:
* Compile: nvcc hello_cuda.cu -o hello_cuda
* Run: ./hello_cuda
---------------------*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_kernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d in block %d!\n", threadIdx.x, blockIdx.x);
}

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    printf("CUDA Hello World Example\n");
    
    // Launch hello kernel with 2 blocks of 4 threads each
    hello_kernel<<<2, 4>>>();
    hipDeviceSynchronize();
    
    printf("\nVector Addition Example\n");
    
    // Simple vector addition
    const int N = 8;
    float h_a[N] = {1, 2, 3, 4, 5, 6, 7, 8};
    float h_b[N] = {8, 7, 6, 5, 4, 3, 2, 1};
    float h_c[N];
    
    float *d_a, *d_b, *d_c;
    
    // Allocate global memory
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    
    // Copy memory to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel with 1 block of 8 threads
    vector_add<<<1, 8>>>(d_a, d_b, d_c, N);
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print results
    for (int i = 0; i < N; i++) {
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
    }
    
    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}